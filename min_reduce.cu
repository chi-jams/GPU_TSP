#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <float.h>

#define uint64_t unsigned long long
#define BLOCK_SIZE 32 
#define blocks_needed(N) ((N / (2 * BLOCK_SIZE)) + (N % (2 * BLOCK_SIZE) == 0 ? 0 : 1))

template <typename T>
__global__ void _d_gen_index_arr(T* d_nums, int N) {
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * BLOCK_SIZE + tid;
    unsigned int gridSize = BLOCK_SIZE * gridDim.x;

    while (i < N) {
        d_nums[i] = i;
        i += gridSize;
    }
}

template <typename T>
T* d_gen_index_arr(int N) {
    T* d_nums; 
    hipMalloc(&d_nums, sizeof(T) * N);

    _d_gen_index_arr<T><<<1, BLOCK_SIZE>>>(d_nums, N);    

    return d_nums;
}

#define min_reduce_it(BLK_SIZE) if (BLOCK_SIZE >= BLK_SIZE) { \
    if (tid < BLK_SIZE / 2 && sdata[tid+BLK_SIZE/2] < sdata[tid]) { \
        sdata[tid] = sdata[tid+BLK_SIZE/2]; \
    } \
    __syncthreads(); \
}

#define min_reduce_warp(BLK_SIZE) if (BLOCK_SIZE >= BLK_SIZE) { \
    if (sdata[tid+BLK_SIZE/2] < sdata[tid]) { \
        sdata[tid] = sdata[tid+BLK_SIZE/2]; \
    } \
    __syncwarp(); \
}

template <typename T>
__global__ void d_min_reduce(const T* d_nums, T* d_res, int N) {
    __shared__ T sdata[2 * BLOCK_SIZE];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * 2 * BLOCK_SIZE + tid;
    unsigned int gridSize = BLOCK_SIZE * 2 * gridDim.x;

    sdata[tid] = DBL_MAX;

    // This rolls all would-be blocks into a single block
    while (i < N) {
        if (d_nums[i] < d_nums[i + BLOCK_SIZE] && d_nums[i] < sdata[tid]) {
            sdata[tid] = d_nums[i];
        }
        else if (d_nums[i + BLOCK_SIZE] < sdata[tid]) {
            sdata[tid] = d_nums[i + BLOCK_SIZE];
        }
        i += gridSize;
    }
    __syncthreads();

    min_reduce_it(512);
    min_reduce_it(256);
    min_reduce_it(128);
    if (tid < 32) {
        min_reduce_warp(64);
        min_reduce_warp(32);
        min_reduce_warp(16);
        min_reduce_warp(8);
        min_reduce_warp(4);
        min_reduce_warp(2);
    }

    if (tid == 0) {
        d_res[blockIdx.x] = sdata[0];
    }
}

// TODO: Finish converting this to min_reduce
template <typename T>
T min_reduce(const T* nums, int N) {
    unsigned int num_blocks = blocks_needed(N);
    T* d_nums;
    T* d_res;
    
    hipMalloc(&d_nums, num_blocks * sizeof(T) * 2 * BLOCK_SIZE);
    hipMalloc(&d_res, num_blocks * sizeof(T));
    hipMemset(d_nums, -1, num_blocks * sizeof(T) * 2 * BLOCK_SIZE);
    hipMemcpy(d_nums, nums, sizeof(T) * N, hipMemcpyHostToDevice);

    // TODO: recursive version, for better GPU utilization
    //d_sum_reduce<T><<<blocks_needed, BLOCK_SIZE>>>(d_nums, d_res, N);
    d_min_reduce<T><<<1, BLOCK_SIZE>>>(d_nums, d_res, N);

    T res; 
    hipMemcpy(&res, d_res, sizeof(T), hipMemcpyDeviceToHost);
    
    hipFree(d_nums);
    hipFree(d_res);

    return res;
}

/* BEGIN: Stuff that should immediately die once we start actually linking
 * things
 */
template <typename T>
T rand_range(T min, T max) {
    double u = rand() / (double)RAND_MAX;
    return (max - min + 1) * u + min;
}

template <typename T>
T* gen_ints(int N) {
    T* nums = (T*) malloc(sizeof(T) * N);
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        nums[i] = rand_range<T>(0, 1000);
    }

    return nums;
}

double* gen_doubles(int N, double min, double max) {
    double* nums = (double*) malloc(sizeof(double) * N);
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        double u = rand() / (double)RAND_MAX;
        nums[i] = (max - min) * u + min;
    }

    return nums;
}
/* END: Stuff that should die */

int main(int argc, char* argv[]) {
    if (argc != 2) {
        printf("Usage: %s N\n", argv[0]);
        exit(-1);
    }

    // Specify the size of the random set of points
    long int N = strtol(argv[1], NULL, 10);
    if (N <= 0) {
        printf("Please enter a positive int for N\n");
        exit(-1);
    }
    else if (N == LONG_MAX || N == LONG_MIN) {
        printf("The provided N is too %s.\n", N == LONG_MAX ? "large" : "small");
        exit(-1);
    }

    printf("N: %d\n", N);

    double* nums = gen_doubles(N, 0, 1000);
    /*
    for (int i = 0; i < N; i++)
        printf("%d: %lf\n", i, nums[i]);
    */

    double min = nums[0];
    int min_ind = 0;
    for (int i = 1; i < N; i++) {
        if (nums[i] < min) {
            min = nums[i];
            min_ind = i;
        }
    }
    printf("Serial min is %lf at %llu\n", min, min_ind);

    double par_min = min_reduce<double>(nums, N);
    printf("Parallel min: %lf\n", par_min);
    
    free(nums);
    return 0;
}

