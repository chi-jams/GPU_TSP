#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>

#define BLOCK_SIZE 32 
#define blocks_needed(N) ((N / (2 * BLOCK_SIZE)) + (N % (2 * BLOCK_SIZE) == 0 ? 0 : 1))

template <typename T>
__global__ void _d_gen_index_arr(const T* d_nums, int N);

template <typename T>
T* d_gen_index_arr(const T* d_nums, int N);

template <typename T>
__global__ void d_sum_reduce(const T* d_nums, T* d_res, int N);

template <typename T>
T sum_reduce(const T* nums, int N);

template <typename T>
T rand_range(T min, T max);

template <typename T>
T* gen_ints(int N);

int main(int argc, char* argv[]) {
    if (argc != 2) {
        printf("Usage: %s N\n", argv[0]);
        exit(-1);
    }

    // Specify the size of the random set of points
    long int N = strtol(argv[1], NULL, 10);
    if (N <= 0) {
        printf("Please enter a positive int for N\n");
        exit(-1);
    }
    else if (N == LONG_MAX || N == LONG_MIN) {
        printf("The provided N is too %s.\n", N == LONG_MAX ? "large" : "small");
        exit(-1);
    }

    printf("N: %d\n", N);

    unsigned long long* nums = gen_ints<unsigned long long>(N);

    /*
    for (int i = 0; i < N; i++)
        printf("%d: %d\n", i, nums[i]);
    */

    unsigned long long sum = 0;
    for (int i = 0; i < N; i++)
        sum += nums[i];
    printf("Serial Sum: %llu\n", sum);

    unsigned long long par_sum = sum_reduce<unsigned long long>(nums, N);
    printf("Parallel Sum: %llu\n", par_sum);

    free(nums);
    return 0;
}

template <typename T>
__global__ void _d_gen_index_arr(const T* d_nums, int N) {
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * BLOCK_SIZE + tid;
    unsigned int gridSize = BLOCK_SIZE * gridDim.x;

    while (i < N) {
        d_nums[i] = i;
        i += gridSize;
    }
}

template <typename T>
T* d_gen_index_arr(int N) {
    T* d_nums; 
    hipMalloc(&d_nums, sizeof(T) * N);

    _d_gen_index_arr(d_nums, N);    

    return d_nums;
}

template <typename T>
__global__ void d_sum_reduce(const T* d_nums, T* d_res, int N) {
    __shared__ T sdata[2 * BLOCK_SIZE];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * 2 * BLOCK_SIZE + tid;
    unsigned int gridSize = BLOCK_SIZE * 2 * gridDim.x;

    sdata[tid] = 0;

    // This layers all would-be blocks into a single block
    while (i < N) {
        sdata[tid] += d_nums[i] + d_nums[i + BLOCK_SIZE];
        i += gridSize;
    }
    __syncthreads();

    if (BLOCK_SIZE >= 512) {if(tid < 256) sdata[tid] += sdata[tid+256];__syncthreads();}
    if (BLOCK_SIZE >= 256) {if(tid < 128) sdata[tid] += sdata[tid+128];__syncthreads();}
    if (BLOCK_SIZE >= 128) {if(tid <  64) sdata[tid] += sdata[tid+ 64];__syncthreads();}
    // below in one warp
    if (tid < 32) {
        if (BLOCK_SIZE >= 64) {sdata[tid] += sdata[tid + 32];__syncwarp();}
        if (BLOCK_SIZE >= 32) {sdata[tid] += sdata[tid + 16];__syncwarp();}
        if (BLOCK_SIZE >= 16) {sdata[tid] += sdata[tid +  8];__syncwarp();}
        if (BLOCK_SIZE >=  8) {sdata[tid] += sdata[tid +  4];__syncwarp();}
        if (BLOCK_SIZE >=  4) {sdata[tid] += sdata[tid +  2];__syncwarp();}
        if (BLOCK_SIZE >=  2) {sdata[tid] += sdata[tid +  1];__syncwarp();}
    }

    if (tid == 0)
        d_res[blockIdx.x] = sdata[0];
}

template <typename T>
T sum_reduce(const T* nums, int N) {
    unsigned int num_blocks = blocks_needed(N);
    T* d_nums;
    T* d_res;
    
    hipMalloc(&d_nums, num_blocks * sizeof(T) * 2 * BLOCK_SIZE);
    hipMalloc(&d_res, num_blocks * sizeof(T));
    hipMemcpy(d_nums, nums, sizeof(T) * N, hipMemcpyHostToDevice);

    // recursive version
    //d_sum_reduce<T><<<num_blocks, BLOCK_SIZE>>>(d_nums, d_res, N);
    d_sum_reduce<T><<<1, BLOCK_SIZE>>>(d_nums, d_res, N);

    T res; 
    hipMemcpy(&res, d_res, sizeof(T), hipMemcpyDeviceToHost);
    
    hipFree(d_nums);
    hipFree(d_res);

    return res;
}

template <typename T>
T rand_range(T min, T max) {
    double u = rand() / (double)RAND_MAX;
    return (max - min + 1) * u + min;
}

template <typename T>
T* gen_ints(int N) {
    T* nums = (T*) malloc(sizeof(T) * N);
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        nums[i] = rand_range<T>(0, 1000);
        //nums[i + 1] = rand_range(0, 1000);
    }

    return nums;
}

